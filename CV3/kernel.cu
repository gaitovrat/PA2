#include "hip/hip_runtime.h"
﻿#include <cudaDefs.h>
#include <time.h>
#include <math.h>
#include <random>

//WARNING!!! Do not change TPB and NO_FORCES for this demo !!!
constexpr unsigned int TPB = 128;
constexpr unsigned int NO_FORCES = 256;
constexpr unsigned int NO_RAIN_DROPS = 1 << 20;

constexpr unsigned int MEM_BLOCKS_PER_THREAD_BLOCK = 8;

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

using namespace std;

__host__ float3* createData(const unsigned int length)
{
    //TODO: Generate float3 vectors. You can use 'make_float3' method.
    float3* data = new float3[length];

    random_device rd;
    uniform_int_distribution<int> dist(0, length);

    for (int i = 0; i < length; ++i)
    {
        data[i].x = dist(rd);
        data[i].y = dist(rd);
        data[i].z = dist(rd);
    }

    return data;
}

__host__ void printData(const float3* data, const unsigned int length)
{
    if (data == 0) return;
    const float3* ptr = data;
    for (unsigned int i = 0; i < length; i++, ptr++)
    {
        printf("%5.2f %5.2f %5.2f ", ptr->x, ptr->y, ptr->z);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>    Sums the forces to get the final one using parallel reduction.
///             WARNING!!! The method was written to meet input requirements of our example, i.e. 128 threads and 256 forces  </summary>
/// <param name="dForces">          The forces. </param>
/// <param name="noForces">       The number of forces. </param>
/// <param name="dFinalForce">    [in,out] If non-null, the final force. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void reduce(const float3* __restrict__ dForces, const unsigned int noForces, float3* __restrict__ dFinalForce)
{
    __shared__ float3 sForces[TPB];                    //SEE THE WARNING MESSAGE !!!
    unsigned int tid = threadIdx.x;
    unsigned int next = TPB;                        //SEE THE WARNING MESSAGE !!!

    //TODO: Make the reduction
    float3* src1 = &sForces[tid];
    float3* src2 = (float3*)&dForces[tid + next];
    volatile float3* vsrc1;
    volatile float3* vsrc2;

    *src1 = dForces[tid];

    while (next != 0)
    {
        if (next <= 32)
        {
            vsrc1 = &sForces[tid];
            vsrc2 = vsrc1 + next;

            vsrc1->x += vsrc2->x;
            vsrc1->y += vsrc2->y;
            vsrc1->z += vsrc2->z;
        }
        else
        {
            src1->x += src2->x;
            src1->y += src2->y;
            src1->z += src2->z;

            __syncthreads();
        }

        next >>= 1;
        src2 = src1 + next;
        if (tid >= next) return;
    }

    if (tid == 0)
    {
        dFinalForce->x = vsrc1->x;
        dFinalForce->y = vsrc1->y;
        dFinalForce->z = vsrc1->z;
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>    Adds the FinalForce to every Rain drops position. </summary>
/// <param name="dFinalForce">    The final force. </param>
/// <param name="noRainDrops">    The number of rain drops. </param>
/// <param name="dRainDrops">     [in,out] If non-null, the rain drops positions. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void add(const float3* __restrict__ dFinalForce, const unsigned int noRainDrops, float3* __restrict__ dRainDrops)
{
    //TODO: Add the FinalForce to every Rain drops position.
    unsigned int tid = threadIdx.x;
    unsigned int size = blockDim.x;

    while (tid < size)
    {
        dRainDrops[tid].x += dFinalForce->x;
        dRainDrops[tid].y += dFinalForce->y;
        dRainDrops[tid].z += dFinalForce->z;

        tid += size;
    }
}


int main(int argc, char* argv[])
{
    initializeCUDA(deviceProp);

    hipEvent_t startEvent, stopEvent;
    float elapsedTime;

    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);

    float3* hForces = createData(NO_FORCES);
    float3* hDrops = createData(NO_RAIN_DROPS);

    float3* dForces = nullptr;
    float3* dDrops = nullptr;
    float3* dFinalForce = nullptr;

    checkCudaErrors(hipMalloc((void**)&dForces, NO_FORCES * sizeof(float3)));
    checkCudaErrors(hipMemcpy(dForces, hForces, NO_FORCES * sizeof(float3), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&dDrops, NO_RAIN_DROPS * sizeof(float3)));
    checkCudaErrors(hipMemcpy(dDrops, hDrops, NO_RAIN_DROPS * sizeof(float3), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&dFinalForce, sizeof(float3)));

    dim3 dimBlock = dim3(TPB, 1);
    dim3 dimGrid = dim3(1, 1);

    for (unsigned int i = 0; i < 1000; i++)
    {
        reduce<<<dimGrid, dimBlock>>>(dForces, NO_FORCES, dFinalForce);
        add<<<dimGrid, dimBlock>> (dFinalForce, NO_RAIN_DROPS, dDrops);
    }

    checkDeviceMatrix<float>((float*)dFinalForce, sizeof(float3), 1, 3, "%5.2f ", "Final force");
    // checkDeviceMatrix<float>((float*)dDrops, sizeof(float3), NO_RAIN_DROPS, 3, "%5.2f ", "Final Rain Drops");

    if (hForces)
        delete[] hForces;
    if (hDrops)
        delete[] hDrops;

    checkCudaErrors(hipFree(dForces));
    checkCudaErrors(hipFree(dDrops));

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    printf("Time to get device properties: %f ms", elapsedTime);
}